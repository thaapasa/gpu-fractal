#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <cutil_inline.h>
#include <cutil_math.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

texture<uchar,  3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>     transferTex; // 1D transfer function texture

typedef struct {
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray {
	float3 o;	// origin
	float3 d;	// direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    int maxSteps = 500;
    float tstep = 0.01f;
    float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
    if (!hit) return;
	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from back to front, accumulating color
    float4 sum = make_float4(0.0f);;
    float t = tfar;
	for(int i=0; i<maxSteps; i++) {		
        float3 pos = eyeRay.o + eyeRay.d*t;
        pos = pos*0.5f+0.5f;    // map position to [0, 1] coordinates

        // read from 3D texture
        float sample = tex3D(tex, pos.x, pos.y, pos.z);

        // lookup in transfer function texture
        float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);

        // accumulate result
        sum = lerp(sum, col, col.w*density);

        t -= tstep;
        if (t < tnear) break;
    }
    sum *= brightness;

    if ((x < imageW) && (y < imageH)) {
        // write output color
        uint i = __umul24(y, imageW) + x;
        d_output[i] = rgbaFloatToInt(sum);
    }
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void initCuda(uchar *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    cutilSafeCall( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );  

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    cutilSafeCall(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

    // create transfer function texture
    float4 transferFunc[] = {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray* d_transferFuncArray;
    cutilSafeCall(hipMallocArray( &d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1)); 
    cutilSafeCall(hipMemcpyToArray( d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray( transferTex, d_transferFuncArray, channelDesc2));
}

extern "C" 
void freeCudaBuffers()
{
    cutilSafeCall(hipFreeArray(d_volumeArray));
    cutilSafeCall(hipFreeArray(d_transferFuncArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH, 
				   float density, float brightness, float transferOffset, float transferScale)
{
	d_render<<<gridSize, blockSize>>>( d_output, imageW, imageH, density, 
										brightness, transferOffset, transferScale);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix) );
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
