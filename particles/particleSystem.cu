#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>

#include "particles_kernel.cu"

extern "C"
{

void cudaInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        hipSetDevice( cutGetMaxGflopsDeviceId() );
    }
}

void cudaGLInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }
}

void allocateArray(void **devPtr, size_t size)
{
    cutilSafeCall(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr)
{
    cutilSafeCall(hipFree(devPtr));
}

void threadSync()
{
    cutilSafeCall(hipDeviceSynchronize());
}

void copyArrayFromDevice(void* host, const void* device, unsigned int vbo, int size)
{   
    if (vbo)
        cutilSafeCall(cudaGLMapBufferObject((void**)&device, vbo));

    cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    
    if (vbo)
        cutilSafeCall(cudaGLUnmapBufferObject(vbo));
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void registerGLBufferObject(uint vbo)
{
    cutilSafeCall(cudaGLRegisterBufferObject(vbo));
    // tell the driver that CUDA only writes to the buffer and OpenGL consumes:
	cutilSafeCall(cudaGLSetBufferObjectMapFlags(vbo, cudaGLMapFlagsWriteDiscard));    
}

void unregisterGLBufferObject(uint vbo)
{
    cutilSafeCall(cudaGLUnregisterBufferObject(vbo));
}

void *mapGLBufferObject(uint vbo)
{
    void *ptr;
    cutilSafeCall(cudaGLMapBufferObject(&ptr, vbo));
    return ptr;
}

void unmapGLBufferObject(uint vbo)
{
    cutilSafeCall(cudaGLUnmapBufferObject(vbo));
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void integrateSystem(float *pos,
                     float *vel,
                     float deltaTime,
                     uint numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    integrate<<< numBlocks, numThreads >>>((float4*)pos,
                                           (float4*)vel,
                                           deltaTime,
                                           numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("integrate kernel execution failed");
}

void calcHash(uint*  gridParticleHash,
              uint*  gridParticleIndex,
              float* pos, 
              int    numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float4 *) pos,
                                           numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");
}

void reorderDataAndFindCellStart(uint*  cellStart,
							     uint*  cellEnd,
							     float* sortedPos,
							     float* sortedVel,
                                 uint*  gridParticleHash,
                                 uint*  gridParticleIndex,
							     float* oldPos,
							     float* oldVel,
							     uint   numParticles,
							     uint   numCells)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // set all cells to empty
	cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
#endif

    uint smemSize = sizeof(uint)*(numThreads+1);
    reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
        cellStart,
        cellEnd,
        (float4 *) sortedPos,
        (float4 *) sortedVel,
		gridParticleHash,
		gridParticleIndex,
        (float4 *) oldPos,
        (float4 *) oldVel,
        numParticles);
    cutilCheckMsg("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
    cutilSafeCall(hipUnbindTexture(oldVelTex));
#endif
}

void collide(float* newVel,
             float* sortedPos,
             float* sortedVel,
             uint*  gridParticleIndex,
             uint*  cellStart,
             uint*  cellEnd,
             uint   numParticles,
             uint   numCells)
{
#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));    
#endif

    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    collideD<<< numBlocks, numThreads >>>((float4*)newVel,
                                          (float4*)sortedPos,
                                          (float4*)sortedVel,
                                          gridParticleIndex,
                                          cellStart,
                                          cellEnd,
                                          numParticles);

    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
    cutilSafeCall(hipUnbindTexture(oldVelTex));
    cutilSafeCall(hipUnbindTexture(cellStartTex));
    cutilSafeCall(hipUnbindTexture(cellEndTex));
#endif
}

}   // extern "C"
